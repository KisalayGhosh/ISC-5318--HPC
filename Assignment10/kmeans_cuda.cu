#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
extern "C" {
    #define STB_IMAGE_IMPLEMENTATION
    #define STB_IMAGE_WRITE_IMPLEMENTATION
    #include "stb_image.h"
    #include "stb_image_write.h"
    }
    



#define BLOCK_SIZE 256

typedef struct {
    float r, g, b;
} Color;
//euclidean distance calc
__device__ float color_distance(Color a, Color b) {
    return sqrtf((a.r - b.r) * (a.r - b.r) +
                 (a.g - b.g) * (a.g - b.g) +
                 (a.b - b.b) * (a.b - b.b));
}

__global__ void assign_clusters(Color* pixels, Color* centroids, int* labels, Color* sums, int* counts, int num_pixels, int k) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_pixels) return;

    float min_dist = 1e20;
    int best_k = 0;

    for (int i = 0; i < k; i++) {
        float dist = color_distance(pixels[idx], centroids[i]);
        if (dist < min_dist) {
            min_dist = dist;
            best_k = i;
        }
    }

    labels[idx] = best_k;
    atomicAdd(&sums[best_k].r, pixels[idx].r);
    atomicAdd(&sums[best_k].g, pixels[idx].g);
    atomicAdd(&sums[best_k].b, pixels[idx].b);
    atomicAdd(&counts[best_k], 1);
}
// Kernel: Update centroid positions by averaging
__global__ void update_centroids(Color* centroids, Color* sums, int* counts, int k) {
    int idx = threadIdx.x;
    if (idx >= k) return;
    if (counts[idx] > 0) {
        centroids[idx].r = sums[idx].r / counts[idx];
        centroids[idx].g = sums[idx].g / counts[idx];
        centroids[idx].b = sums[idx].b / counts[idx];
    }
}
// Kernel: Assign final pixel color based on final centroid
__global__ void color_pixels(Color* pixels, Color* centroids, int* labels, int num_pixels) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_pixels) return;
    int label = labels[idx];
    pixels[idx] = centroids[label];
}

int main(int argc, char* argv[]) {
    if (argc < 5) {
        printf("Usage: %s input.jpg output.jpg k iterations\n", argv[0]);
        return 1;
    }

    int width, height, channels;
    unsigned char* img_data = stbi_load(argv[1], &width, &height, &channels, 3);
    if (!img_data) {
        fprintf(stderr, "Failed to load image.\n");
        return 1;
    }
// Convert image data to Color struct array
    int num_pixels = width * height;
    Color* h_pixels = (Color*)malloc(sizeof(Color) * num_pixels);
    for (int i = 0; i < num_pixels; i++) {
        h_pixels[i].r = img_data[i * 3];
        h_pixels[i].g = img_data[i * 3 + 1];
        h_pixels[i].b = img_data[i * 3 + 2];
    }

    int k = atoi(argv[3]);
    int iterations = atoi(argv[4]);

    Color *d_pixels, *d_centroids, *d_sums;
    int *d_labels, *d_counts;
    hipMalloc(&d_pixels, sizeof(Color) * num_pixels);
    hipMalloc(&d_labels, sizeof(int) * num_pixels);
    hipMalloc(&d_centroids, sizeof(Color) * k);
    hipMalloc(&d_sums, sizeof(Color) * k);
    hipMalloc(&d_counts, sizeof(int) * k);

    hipMemcpy(d_pixels, h_pixels, sizeof(Color) * num_pixels, hipMemcpyHostToDevice);
        // Initialize centroids evenly spaced from image pixels
    Color* h_centroids = (Color*)malloc(sizeof(Color) * k);
    for (int i = 0; i < k; i++) h_centroids[i] = h_pixels[i * num_pixels / k];
    hipMemcpy(d_centroids, h_centroids, sizeof(Color) * k, hipMemcpyHostToDevice);

    int blocks = (num_pixels + BLOCK_SIZE - 1) / BLOCK_SIZE;

    float elapsed;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    for (int it = 0; it < iterations; it++) {
        hipMemset(d_sums, 0, sizeof(Color) * k);
        hipMemset(d_counts, 0, sizeof(int) * k);
        //Assign clusters
        assign_clusters<<<blocks, BLOCK_SIZE>>>(d_pixels, d_centroids, d_labels, d_sums, d_counts, num_pixels, k);
        update_centroids<<<1, k>>>(d_centroids, d_sums, d_counts, k);
    }

    color_pixels<<<blocks, BLOCK_SIZE>>>(d_pixels, d_centroids, d_labels, num_pixels);

    hipMemcpy(h_pixels, d_pixels, sizeof(Color) * num_pixels, hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);
    printf("CUDA Execution Time: %.4f ms\n", elapsed);

    for (int i = 0; i < num_pixels; i++) {
        img_data[i * 3]     = (unsigned char)h_pixels[i].r;
        img_data[i * 3 + 1] = (unsigned char)h_pixels[i].g;
        img_data[i * 3 + 2] = (unsigned char)h_pixels[i].b;
    }
    stbi_write_jpg(argv[2], width, height, 3, img_data, 100);

    free(h_pixels);
    free(h_centroids);
    stbi_image_free(img_data);
    hipFree(d_pixels);
    hipFree(d_labels);
    hipFree(d_centroids);
    hipFree(d_sums);
    hipFree(d_counts);

    return 0;
}
